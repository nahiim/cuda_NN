﻿#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <>

#include "kernels.h"

const int input_size = 2;
const int hidden_size = 4;
const int output_size = 1;
const int epochs = 20000;
const float learning_rate = 0.05f;

const float h_inputs[4][2] = { {0,0}, {0,1}, {1,0}, {1,1} };
const float h_targets[4] = { 0, 1, 1, 0 };

float* d_inputs, * d_hidden, * d_output, * d_target;
float* d_weights1, * d_bias1, * d_weights2, * d_bias2;


int main()
{
    hipMalloc(&d_inputs, sizeof(float) * input_size);
    hipMalloc(&d_hidden, sizeof(float) * hidden_size);
    hipMalloc(&d_output, sizeof(float));
    hipMalloc(&d_target, sizeof(float));
    hipMalloc(&d_weights1, sizeof(float) * input_size * hidden_size);
    hipMalloc(&d_bias1, sizeof(float) * hidden_size);
    hipMalloc(&d_weights2, sizeof(float) * hidden_size * output_size);
    hipMalloc(&d_bias2, sizeof(float) * output_size);

    float h_w1[input_size * hidden_size];
    float h_b1[hidden_size] = {};
    float h_w2[hidden_size * output_size];
    float h_b2[output_size] = {};
    srand(42);
    for (int i = 0; i < input_size * hidden_size; i++) h_w1[i] = (rand() / (float)RAND_MAX - 0.5f);
    for (int i = 0; i < hidden_size * output_size; i++) h_w2[i] = (rand() / (float)RAND_MAX - 0.5f);

    hipMemcpy(d_weights1, h_w1, sizeof(h_w1), hipMemcpyHostToDevice);
    hipMemcpy(d_bias1, h_b1, sizeof(h_b1), hipMemcpyHostToDevice);
    hipMemcpy(d_weights2, h_w2, sizeof(h_w2), hipMemcpyHostToDevice);
    hipMemcpy(d_bias2, h_b2, sizeof(h_b2), hipMemcpyHostToDevice);


    // Training
    for (int epoch = 0; epoch < epochs; epoch++)
    {
        for (int i = 0; i < 4; i++)
        {
            hipMemcpy(d_inputs, h_inputs[i], sizeof(float) * input_size, hipMemcpyHostToDevice);
            hipMemcpy(d_target, &h_targets[i], sizeof(float), hipMemcpyHostToDevice);

            forwardPass << <1, hidden_size >> > (d_inputs, d_weights1, d_bias1, d_hidden, input_size, hidden_size);
            hipDeviceSynchronize();

            forwardPass<< <1, output_size >> > (d_hidden, d_weights2, d_bias2, d_output, hidden_size, output_size);
            hipDeviceSynchronize();

            backPropagation << <1, hidden_size >> > (d_inputs, d_hidden, d_output,
                d_weights1, d_bias1, d_weights2, d_bias2,
                d_target,
                input_size, hidden_size, learning_rate);
            hipDeviceSynchronize();
        }
    }

    // Inference
    printf("=== Inference After Training ===\n");
    for (int i = 0; i < 4; i++)
    {
        hipMemcpy(d_inputs, h_inputs[i], sizeof(float) * input_size, hipMemcpyHostToDevice);
        forwardPass<< <1, hidden_size >> > (d_inputs, d_weights1, d_bias1, d_hidden, input_size, hidden_size);
        hipDeviceSynchronize();
        forwardPass<< <1, output_size >> > (d_hidden, d_weights2, d_bias2, d_output, hidden_size, output_size);
        hipDeviceSynchronize();

        float out;
        hipMemcpy(&out, d_output, sizeof(float), hipMemcpyDeviceToHost);
        printf("%d XOR %d = %.4f (target=%.1f)\n",
            (int)h_inputs[i][0], (int)h_inputs[i][1], out, h_targets[i]);
    }

    hipFree(d_inputs); hipFree(d_hidden); hipFree(d_output); hipFree(d_target);
    hipFree(d_weights1); hipFree(d_bias1); hipFree(d_weights2); hipFree(d_bias2);
    
    
    return 0;
}